#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <sstream>
#include <fstream>

#define BLOCK_DIM 1024
#define MAX_DIMS 10
#define TILE 32

#define ADD_FUNC       1
#define MUL_FUNC       2
#define ID_FUNC        3
#define NEG_FUNC       4
#define LT_FUNC        5
#define EQ_FUNC        6
#define SIGMOID_FUNC   7
#define RELU_FUNC      8
#define RELU_BACK_FUNC 9
#define LOG_FUNC       10
#define LOG_BACK_FUNC  11
#define EXP_FUNC       12
#define INV_FUNC       13
#define INV_BACK_FUNC  14
#define IS_CLOSE_FUNC  15
#define MAX_FUNC       16
#define POW            17
#define TANH           18

__device__ float fn(int fn_id, float x, float y=0) {
    switch(fn_id) {
      case ADD_FUNC: {
        return x + y;
      }
      case MUL_FUNC: {
        return x * y;
      }
      case ID_FUNC: {
      	return x;
      }
      case NEG_FUNC: {
        return -x;
      }
      case LT_FUNC: {
        if (x < y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case EQ_FUNC: {
        if (x == y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case SIGMOID_FUNC: {
        if (x >= 0) {
          return 1.0 / (1.0 + exp(-x));
        }
        else {
          return exp(x) / (1.0 + exp(x));
        }
      }
      case RELU_FUNC: {
        return max(x, 0.0);
      }
      case RELU_BACK_FUNC: {
        if (x > 0) {
          return y;
        }
        else {
          return 0.0;
        }
      }
      case LOG_FUNC: {
        return log(x + 1e-6);
      }
      case LOG_BACK_FUNC: {
        return y / (x + 1e-6);
      }
      case EXP_FUNC: {
        return exp(x);
      }
      case INV_FUNC: {
        return float(1.0 / x);
      }
      case INV_BACK_FUNC: {
        return -(1.0 / (x * x)) * y;
      }
      case IS_CLOSE_FUNC: {
        return (x - y < 1e-2) && (y - x < 1e-2);
      }
      case MAX_FUNC: {
        if (x > y) {
          return x;
        }
        else {
          return y;
        }
      }
      case POW: {
        return pow(x, y);
      }
      case TANH: {
        return tanh(x);
      }
      default: {
        return x + y;
      }
    }
    
}


__device__ int index_to_position(const int* index, const int* strides, int num_dims) {
  /**
   * Converts a multidimensional tensor index into a single-dimensional position in storage
   * based on strides.
   * Args:
   *    index: index tuple of ints
   *    strides: tensor strides
   *    num_dims: number of dimensions in the tensor, e.g. shape/strides of [2, 3, 4] has 3 dimensions
   *
   * Returns:
   *    int - position in storage
  */
    int position = 0;
    for (int i = 0; i < num_dims; ++i) {
        position += index[i] * strides[i];
    }
    return position;
}

__device__ void to_index(int ordinal, const int* shape, int* out_index, int num_dims) {
  /**
   * Convert an ordinal to an index in the shape. Should ensure that enumerating position 0 ... size of
   * a tensor produces every index exactly once. It may not be the inverse of index_to_position.
   * Args:
   *    ordinal: ordinal position to convert
   *    shape: tensor shape
   *    out_index: return index corresponding to position
   *    num_dims: number of dimensions in the tensor
   *
   * Returns:
   *    None (Fills in out_index)
   */
    int cur_ord = ordinal;
    for (int i = num_dims - 1; i >= 0; --i) {
        int sh = shape[i];
        out_index[i] = cur_ord % sh;
        cur_ord /= sh;
    }
}

__device__ void broadcast_index(const int* big_index, const int* big_shape, const int* shape, int* out_index, int num_dims_big, int num_dims) {
  /**
   * Convert a big_index into big_shape to a smaller out_index into shape following broadcasting rules.
   * In this case it may be larger or with more dimensions than the shape given.
   * Additional dimensions may need to be mapped to 0 or removed.
   *
   * Args:
   *    big_index: multidimensional index of bigger tensor
   *    big_shape: tensor shape of bigger tensor
   *    shape: tensor shape of smaller tensor
   *    out_index: multidimensional index of smaller tensor
   *    nums_big_dims: number of dimensions in bigger tensor
   *    num_dims: number of dimensions in smaller tensor
   *
   * Returns:
   *    None (Fills in out_index)
  */
    for (int i = 0; i < num_dims; ++i) {
        if (shape[i] > 1) {
            out_index[i] = big_index[i + (num_dims_big - num_dims)];
        } else {
            out_index[i] = 0;
        }
    }
}


__global__ void MatrixMultiplyKernel(
    float* out,
    const int* out_shape,
    const int* out_strides,
    float* a_storage,
    const int* a_shape,
    const int* a_strides,
    float* b_storage,
    const int* b_shape,
    const int* b_strides
) {
  /**
   * Multiply two (compact) matrices into an output (also compact) matrix. Matrix a and b are both in a batch
   * format, with shape [batch_size, m, n], [batch_size, n, p].
   * Requirements:
   * - All data must be first moved to shared memory.
   * - Only read each cell in a and b once.
   * - Only write to global memory once per kernel.
   * There is guarantee that a_shape[0] == b_shape[0], a_shape[2] == b_shape[1],
   * and out_shape[0] == a_shape[0], out_shape[1] == b_shape[1]
   *
   * Args:
   *   out: compact 1D array of size batch_size x m x p to write the output to
   *   out_shape: shape of the output array
   *   out_strides: strides of the output array
   *   a_storage: compact 1D array of size batch_size x m x n
   *   a_shape: shape of the a array
   *   a_strides: strides of the a array
   *   b_storage: comapct 2D array of size batch_size x n x p
   *   b_shape: shape of the b array
   *   b_strides: strides of the b array
   *
   * Returns:
   *   None (Fills in out array)
   */

    __shared__ float a_shared[TILE][TILE];
    __shared__ float b_shared[TILE][TILE];

    // In each block, we will compute a batch of the output matrix
    // All the threads in the block will work together to compute this batch
    int batch = blockIdx.z;
    int a_batch_stride = a_shape[0] > 1 ? a_strides[0] : 0;
    int b_batch_stride = b_shape[0] > 1 ? b_strides[0] : 0;


    /// BEGIN ASSIGN1_2
    /// TODO
    // Hints:
    // 1. Compute the row and column of the output matrix this block will compute
    // 2. Compute the position in the output array that this thread will write to
    // 3. Iterate over tiles of the two input matrices, read the data into shared memory
    // 4. Synchronize to make sure the data is available to all threads
    // 5. Compute the output tile for this thread block
    // 6. Synchronize to make sure all threads are done computing the output tile for (row, col)
    // 7. Write the output to global memory

    //Task1
    
    int bx = blockIdx.x;  int by = blockIdx.y;
    int ty = threadIdx.x;  int tx = threadIdx.y;

    int col = blockDim.y*by + tx;
    int row = blockDim.x*bx + ty;
    
    int ndims = 3;

    //Task3
    int width = a_shape[2];
    
    float result = 0;
    for(int ph = 0; ph<(width-1+TILE)/TILE; ++ph){
      int a_index[3] = {batch, row, ph*TILE + tx};
      int b_index[3] = {batch, ph*TILE + ty, col};
      int a_pos = index_to_position(a_index, a_strides, ndims);
      int b_pos = index_to_position(b_index, b_strides, ndims);

      a_shared[ty][tx] = a_storage[a_pos];
      b_shared[ty][tx] = b_storage[b_pos];
      __syncthreads();
      for(int k=0; k<TILE; k++){
        result += a_shared[ty][k]*b_shared[k][tx];
      }
      __syncthreads();
    }
  if (row>=out_shape[1] or col>=out_shape[2]) return;

    int out_index[3] = {batch, row, col};
    int out_pos = index_to_position(out_index, out_strides, ndims);
    out[out_pos] = result;

    //assert(false && "Not Implemented");
    /// END ASSIGN1_2
}

__global__ void mapKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int shape_size,
    int fn_id
) {
  /**
   * Map function. Apply a unary function to each element of the input array and store the result in the output array.
   * Optimization: Parallelize over the elements of the output array.
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   * - broadcast_index: converts an index in a smaller array to an index in a larger array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  in_storage: compact 1D array of size in_size
   *  in_shape: shape of the input array
   *  in_strides: strides of the input array
   *  shape_size: number of dimensions in the input and output arrays, assume dimensions are the same
   *  fn_id: id of the function to apply to each element of the input array
   *
   * Returns:
   *  None (Fills in out array)
   */

    int out_index[MAX_DIMS];
    int in_index[MAX_DIMS];
    
    /// BEGIN ASSIGN1_2
    /// TODO
    // Hints:
    // 1. Compute the position in the output array that this thread will write to
    // 2. Convert the position to the out_index according to out_shape
    // 3. Broadcast the out_index to the in_index according to in_shape (optional in some cases)
    // 4. Calculate the position of element in in_array according to in_index and in_strides
    // 5. Calculate the position of element in out_array according to out_index and out_strides
    // 6. Apply the unary function to the input element and write the output to the out memory
    

    //Task 1
    int position = blockIdx.x*blockDim.x + threadIdx.x;
    if (position >= out_size) return;

    to_index(position, out_shape, out_index, shape_size);
    //Task 3
    broadcast_index(out_index, out_shape, in_shape, in_index, shape_size, shape_size);
    //Task 4
    int in_position = index_to_position(in_index, in_strides, shape_size);
    //Task 5
    // int out_position = index_to_position(out_index, out_strides, shape_size);
    //Task 6
    out[position] = fn(fn_id, in_storage[in_position]);
    /// END ASSIGN1_2
}


__global__ void reduceKernel(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    int reduce_dim,
    float reduce_value,
    int shape_size,
    int fn_id
) {
  /**
   * Reduce function. Apply a reduce function to elements of the input array a and store the result in the output array.
   * Optimization:
   * Parallelize over the reduction operation. Each kernel performs one reduction.
   * e.g. a = [[1, 2, 3], [4, 5, 6]], kernel0 computes reduce([1, 2, 3]), kernel1 computes reduce([4, 5, 6]).
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  a_storage: compact 1D array of size in_size
   *  a_shape: shape of the input array
   *  a_strides: strides of the input array
   *  reduce_dim: dimension to reduce on
   *  reduce_value: initial value for the reduction
   *  shape_size: number of dimensions in the input & output array, assert dimensions are the same
   *  fn_id: id of the reduce function, currently only support add, multiply, and max
   *
   *
   * Returns:
   *  None (Fills in out array)
   */

    // __shared__ double cache[BLOCK_DIM]; // Uncomment this line if you want to use shared memory to store partial results
    int out_index[MAX_DIMS];

    /// BEGIN ASSIGN1_2
    /// TODO
    // 1. Define the position of the output element that this thread or this block will write to
    int out_pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_pos >= out_size) return;

    // 2. Convert the out_pos to the out_index according to out_shape
    to_index(out_pos, out_shape, out_index, shape_size);

    // 3. Initialize the reduce_value to the output element
    float result = reduce_value;
    //Check this
    // 4. Iterate over the reduce_dim dimension of the input array to compute the reduced value
    for (int i = 0; i < a_shape[reduce_dim]; ++i) {
        out_index[reduce_dim] = i;
        int a_position = index_to_position(out_index, a_strides, shape_size);
        result = fn(fn_id, result, a_storage[a_position]);
    }

    // 5. Write the reduced value to out memory
    out[out_pos] = result;
    /// END ASSIGN1_2
}

__global__ void zipKernel(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    int out_shape_size,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_shape_size,
    int fn_id
) {
  /**
   * Zip function. Apply a binary function to elements of the input array a & b and store the result in the output array.
   * Optimization: Parallelize over the elements of the output array.
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   * - broadcast_index: converts an index in a smaller array to an index in a larger array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  out_shape_size: number of dimensions in the output array
   *  a_storage: compact 1D array of size in_size
   *  a_shape: shape of the input array
   *  a_strides: strides of the input array
   *  a_shape_size: number of dimensions in the input array
   *  b_storage: compact 1D array of size in_size
   *  b_shape: shape of the input array
   *  b_strides: strides of the input array
   *  b_shape_size: number of dimensions in the input array
   *  fn_id: id of the function to apply to each element of the a & b array
   *
   *
   * Returns:
   *  None (Fills in out array)
   */

    int out_index[MAX_DIMS];
    int a_index[MAX_DIMS];
    int b_index[MAX_DIMS];
    //Zip function
    /// BEGIN ASSIGN1_2
    /// TODO
    // Hints:
    // 1. Compute the position in the output array that this thread will write to
    // 2. Convert the position to the out_index according to out_shape
    // 3. Calculate the position of element in out_array according to out_index and out_strides
    // 4. Broadcast the out_index to the a_index according to a_shape
    // 5. Calculate the position of element in a_array according to a_index and a_strides
    // 6. Broadcast the out_index to the b_index according to b_shape
    // 7.Calculate the position of element in b_array according to b_index and b_strides
    // 8. Apply the binary function to the input elements in a_array & b_array and write the output to the out memory
    
    //Task1
    int thread_pos = blockIdx.x*blockDim.x + threadIdx.x;
    if (thread_pos>=out_size) return;
    //Task2
    to_index(thread_pos, out_shape, out_index, out_shape_size);
    //Task3
    int out_pos = index_to_position(out_index, out_strides, out_shape_size);
    //Task4
    broadcast_index(out_index, out_shape, a_shape, a_index, out_shape_size, a_shape_size);
    //Task5
    int a_pos = index_to_position(a_index, a_strides, a_shape_size);
    //Task6
    broadcast_index(out_index, out_shape, b_shape, b_index, out_shape_size, b_shape_size);
    //Task7
    int b_pos = index_to_position(b_index, b_strides, b_shape_size);
    //Task8
    out[out_pos] = fn(fn_id, a_storage[a_pos], b_storage[b_pos]);
    //assert(false && "Not Implemented");
    /// END ASSIGN1_2
}


extern "C" {

void MatrixMultiply(
    float* out,
    int* out_shape,
    int* out_strides,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int batch, int m, int p
) {
    int n = a_shape[2];

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, batch * m * n * sizeof(float));
    hipMalloc(&d_b, batch * n * p * sizeof(float));
    hipMalloc(&d_out, batch * m * p * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, 3 * sizeof(int));
    hipMalloc(&d_out_strides, 3 * sizeof(int));
    hipMalloc(&d_a_shape, 3 * sizeof(int));
    hipMalloc(&d_a_strides, 3 * sizeof(int));
    hipMalloc(&d_b_shape, 3 * sizeof(int));
    hipMalloc(&d_b_strides, 3 * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, batch * m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, batch * n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, 3 * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 32;
    dim3 blockDims(threadsPerBlock, threadsPerBlock, 1); // Adjust these values based on your specific requirements
    dim3 gridDims((m + threadsPerBlock - 1) / threadsPerBlock, (p + threadsPerBlock - 1) / threadsPerBlock, batch);
    MatrixMultiplyKernel<<<gridDims, blockDims>>>(
        d_out, d_out_shape, d_out_strides, d_a, d_a_shape, d_a_strides, d_b, d_b_shape, d_b_strides
    );

    // Copy back to the host
    hipMemcpy(out, d_out, batch * m * p * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Matmul Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}

void tensorMap(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int in_size,
    int shape_size,
    int fn_id
) {
    float *d_out, *d_in;
    // Allocate device memory
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_in, in_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_in_shape, *d_in_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_in_shape, shape_size * sizeof(int));
    hipMalloc(&d_in_strides, shape_size * sizeof(int));

    // Copy data from CPU(host) to GPU(device)
    hipMemcpy(d_in, in_storage, in_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_shape, in_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_strides, in_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 32;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    mapKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, 
      d_in, d_in_shape, d_in_strides, 
      shape_size, fn_id);
    
    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Map Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_in_shape);
    hipFree(d_in_strides);
}


void tensorZip(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_size,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_size,
    int b_shape_size,
    int fn_id
) {
    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, a_size * sizeof(float));
    hipMalloc(&d_b, b_size * sizeof(float));
    hipMalloc(&d_out, out_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, out_shape_size * sizeof(int));
    hipMalloc(&d_out_strides, out_shape_size * sizeof(int));
    hipMalloc(&d_a_shape, a_shape_size * sizeof(int));
    hipMalloc(&d_a_strides, a_shape_size * sizeof(int));
    hipMalloc(&d_b_shape, b_shape_size * sizeof(int));
    hipMalloc(&d_b_strides, b_shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, b_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, b_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, b_shape_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 32;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    zipKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, out_shape_size,
      d_a, d_a_shape, d_a_strides, a_shape_size,
      d_b, d_b_shape, d_b_strides, b_shape_size,
      fn_id);

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();


    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Zip Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}



void tensorReduce(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim, 
    float reduce_value,
    int shape_size,
    int fn_id
) {
    // Allocate device memory
    int a_size = out_size * a_shape[reduce_dim];
    float *d_out, *d_a;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_a, a_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_a_shape, shape_size * sizeof(int));
    hipMalloc(&d_a_strides, shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    // Launch kernel
    int threadsPerBlock = 32;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    reduceKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_out, d_out_shape, d_out_strides, out_size, 
        d_a, d_a_shape, d_a_strides, 
        reduce_dim, reduce_value, shape_size, fn_id
    );
    
    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Reduce Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
}

}
